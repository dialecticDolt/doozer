#include "hip/hip_runtime.h"
#include "sleep.h"
#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>

#ifdef DOOZER_ENABLE_NVTX
#include <nvtx3/nvtx3.hpp>
#endif

using namespace std;
using namespace chrono;

#include <cstdint>

__device__ void gpu_sleep_impl(uint64_t sleep_cycles) {
  unsigned long start = clock64();
  unsigned long cycles_elapsed;
  do {
    cycles_elapsed = clock64() - start;
  } while (cycles_elapsed < sleep_cycles);
}

__global__ void gpu_sleep_kernel(clock_t clock_count) {
  gpu_sleep_impl(clock_count);
}

void gpu_busy_sleep(const int device, const uint64_t cycles,
                    intptr_t stream_ptr) {
#ifdef DOOZER_ENABLE_NVTX
  nvtx3::scoped_range r{"cpp::gpu_busy_sleep"};
#endif
  hipSetDevice(device);
  hipStream_t stream = reinterpret_cast<hipStream_t>(stream_ptr);
  gpu_sleep_kernel<<<1, 1, 0, stream>>>(cycles);
}
